// Filter Analyzer
// Phil Alcorn
// September 18, 2025

// nvcc main.cu -o temp


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define SLOPE -3.010299956639811952137388947244930267681898814621085413104


/***** ***** ***** STRUCTS ***** ***** *****/
typedef struct  
{
	float r_main=0;
	float resistor[6] = {0, 0, 0, 0, 0, 0};
	float capacitor[6] = {0, 0, 0, 0, 0, 0};

} Circuit;
/***** ***** ***** GLOBALS ***** ***** *****/

float* RESISTORS_CPU;
float* CAPACITORS_CPU;

float* RESISTORS_GPU;
float* CAPACITORS_GPU;

int NUM_TEST_FREQUENCIES = 1000;
float MIN_TEST_FREQUENCY = 10;
float MAX_TEST_FREQUENCY = 40000;

const int THREAD_WIDTH = 32;
const int THREAD_HEIGHT =32;
dim3 grid_size;
dim3 block_size;

/***** ***** ***** FUNCITON PROTOTYPES ***** ***** *****/
int get_length_f(float* array);
int fill_array_f(float** destination, const char* file);
void print_array_f(float* arr, int length);
int generate_test_frequencies(float** freq_array, 
							  int num_tests, 
							  float f_min, 
							  float f_max);

// Returns the slope 
float  __device__ evaluate(Circuit c);

void __global__ set_up_devices();


int main() 
{

	int length = fill_array_f(&RESISTORS_CPU, "resistors.txt");
	// print_array_f(RESISTORS_CPU, length);
	
	printf("\n");

	length = fill_array_f(&CAPACITORS_CPU, "capacitors.txt");
	// print_array_f(CAPACITORS_CPU, length);

	float* test_frequencies;

	length = generate_test_frequencies(&test_frequencies, 
									   NUM_TEST_FREQUENCIES, 
									   MIN_TEST_FREQUENCY, 
									   MAX_TEST_FREQUENCY);

	//print_array_f(test_frequencies, length);

	free(test_frequencies);

	free(RESISTORS_CPU);
	free(CAPACITORS_CPU);
	hipFree(RESISTORS_GPU);
	hipFree(CAPACITORS_GPU);
}


int get_length_f(float* array) {  return sizeof(array)/sizeof(array[0]);  };

int fill_array_f(float** destination, const char* file) 
{
	FILE *file_pointer = fopen(file, "r");
	if (!file_pointer) {  perror("fopen"); exit(1);  }
	
	char line[64];
	int capacity =16; // initial size
	int count = 0;
	float *arr = (float*)malloc(capacity * sizeof(float));
	if (!arr) {  exit(1);  }


	while (fgets(line, sizeof(line), file_pointer) != NULL) 
	{
		// Assign the array if necessary
		if (count > capacity) 
		{
			capacity += 16;
			float* tmp = (float*)realloc(arr, capacity * sizeof(float));
			arr = tmp;
		}

		arr[count] = strtof(line, NULL);
		count++;
	}
	// Set the address of our destination pointer to be the address of the array
	// we just created
	*destination = arr; 
	return count;
}

void print_array_f(float* arr, int length) 
{
	for (int i = 0; i < length; i++ )
	{
		printf("Position: %d, Value: %f\n", i, arr[i]);
	}
}


// Need to map the threads to the capacitors in the y direction 
// and the resistors in the x direction. 
//
// The formula for mapping the components is as follows:
// x = threadIdx.x, length is number of discrete resistor values.
// R0 = x % length
// R1 = (x/length) % length
// R2 = (x/length^2) % length
// R2 = (x/length^3) % length
//
void __global__ set_up_devices(dim3* gs, dim3 bs, int c_length, int r_length) 
{
	// Set up number of stuff 
}

float __device__ evaluate(float* resistors, 
						  int r_length, 
						  float* capacitors, 
						  int c_length) 
{
	
	float slope = 0;
	Circuit c;
	c.r_main=10000;
	#pragma unroll 10
	for (int i=0; i<r_length; i++)
	{
		c.resistor[i] = threadIdx.x % r_length;
	}
	return slope;
}


// Want to generate a logarithmically spaced array of frequences 
// from 10Hz to 4kHz (a decade before to a decade after)
int generate_test_frequencies(float** freq_array, 
							  int num_tests, 
							  float f_min, 
							  float f_max)
{
	float* arr = (float*)malloc(sizeof(float) * num_tests);
	for (int i =0; i< num_tests; i++)
	{
		// Make a test frequency
		arr[i] = f_min * pow((f_max/f_min), (float)i/(float)(num_tests-1));
	}

	*freq_array = arr;
	return num_tests;	
}






